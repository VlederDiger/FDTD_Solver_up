#include "hip/hip_runtime.h"
/*
 * UpdaterHz1D.cpp
 *
 *  Created on: 20 сен. 2019 г.
 *      Author: vladislav
 */

#include "UpdaterIntensityTE.h"

__device__
void UpdaterIntensityTE::operator() (const int indx) {
	#define Ex(M, N) Ex[(M) * (gridSizeY) + (N)]
	#define Ey(M, N) Ey[(M) * (gridSizeY) + (N)] // gridSizeY-1?
	const int n = indx % sizeY;
	const int m = indx / sizeY;
	intensity[indx] = intensity[indx] + sqrt(Ex(firstX + m*stepX, firstY + n*stepX)*Ex(firstX + m*stepX, firstY + n*stepX)+Ey(firstX + m*stepX, firstY + n*stepX)*Ey(firstX + m*stepX, firstY + n*stepX));
}
